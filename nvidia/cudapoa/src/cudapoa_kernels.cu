#include "hip/hip_runtime.h"
// Implementation file for CUDA POA kernels.

#include "cudapoa_kernels.cuh"
#include "cudapoa_nw.cu"
#include "cudapoa_topsort.cu"
#include "cudapoa_add_alignment.cu"
#include "cudapoa_generate_consensus.cu"

#include <stdio.h>

namespace nvidia {

namespace cudapoa {

/**
 * @brief The main kernel that runs the partial order alignment
 *        algorithm.
 *
 * @param[out] consensus_d                Device buffer for generated consensus
 * @param[out] coverage_d                 Device buffer for coverage of each base in consensus
 * @param[in] sequences_d                 Device buffer with sequences for all windows
 * @param[in] sequence_lengths_d          Device buffer sequence lengths
 * @param[in] window_details_d            Device buffer with structs 
 *                                        encapsulating sequence details per window
 * @param[in] total_window                Total number of windows to process
 * @param[in] scores                      Device scratch space that scores alignment matrix score
 * @param[in] alignment_graph_d           Device scratch space for backtrace alignment of graph
 * @param[in] alignment_read_d            Device scratch space for backtrace alignment of sequence
 * @param[in] nodes                       Device scratch space for storing unique nodes in graph
 * @param[in] incoming_edges              Device scratch space for storing incoming edges per node
 * @param[in] incoming_edges_count        Device scratch space for storing number of incoming edges per node
 * @param[in] outgoing_edges              Device scratch space for storing outgoing edges per node
 * @param[in] outgoing_edges_count        Device scratch space for storing number of outgoing edges per node
 * @param[in] incoming_edge_w             Device scratch space for storing weight of incoming edges
 * @param[in] outgoing_edge_w             Device scratch space for storing weight of outgoing edges
 * @param[in] sorted_poa                  Device scratch space for storing sorted graph
 * @param[in] node_id_to_pos              Device scratch space for mapping node ID to position in graph
 * @graph[in] node_alignments             Device scratch space for storing alignment nodes per node in graph
 * @param[in] node_alignment_count        Device scratch space for storing number of aligned nodes
 * @param[in] sorted_poa_local_edge_count Device scratch space for maintaining edge counts during topological sort
 * @param[in] consensus_scores            Device scratch space for storing score of each node while traversing graph during consensus
 * @param[in] consensus_predecessors      Device scratch space for storing predecessors of nodes while traversing graph during consensus
 * @param[in] node_marks_d_               Device scratch space for storing node marks when running spoa accurate top sort
 * @param[in] check_aligned_nodes_d_      Device scratch space for storing check for aligned nodes
 * @param[in] nodes_to_visit_d_           device scratch space for storing stack of nodes to be visited in topsort
 * @param[in] node_coverage_counts_d_     device scratch space for storing coverage of each node in graph.
 */
__global__
void generatePOAKernel(uint8_t* consensus_d,
                       uint16_t* coverage_d,
                       uint8_t* sequences_d,
                       uint16_t * sequence_lengths_d,
                       nvidia::cudapoa::WindowDetails * window_details_d,
                       uint32_t total_windows,
                       int16_t* scores_d,
                       int16_t* alignment_graph_d,
                       int16_t* alignment_read_d,
                       uint8_t* nodes_d,
                       uint16_t* incoming_edges_d,
                       uint16_t* incoming_edge_count_d,
                       uint16_t* outgoing_edges_d,
                       uint16_t* outgoing_edge_count_d,
                       uint16_t* incoming_edge_w_d,
                       uint16_t* outgoing_edge_w_d,
                       uint16_t* sorted_poa_d,
                       uint16_t* node_id_to_pos_d,
                       uint16_t* node_alignments_d,
                       uint16_t* node_alignment_count_d,
                       uint16_t* sorted_poa_local_edge_count_d,
                       int32_t* consensus_scores_d,
                       int16_t* consensus_predecessors_d,
                       uint8_t* node_marks_d_,
                       bool* check_aligned_nodes_d_,
                       uint16_t* nodes_to_visit_d_,
                       uint16_t* node_coverage_counts_d_)
{

    uint32_t block_idx = blockIdx.x;
    uint32_t thread_idx = threadIdx.x;

    long long int back_time = 0;
    long long int nw_time = 0;
    long long int add_time = 0;
    long long int top_time = 0;

    if (block_idx > total_windows)
        return;

    // Find the buffer offsets for each thread within the global memory buffers.
    uint8_t* nodes = &nodes_d[CUDAPOA_MAX_NODES_PER_WINDOW * block_idx];
    uint16_t* incoming_edges = &incoming_edges_d[block_idx * CUDAPOA_MAX_NODES_PER_WINDOW * CUDAPOA_MAX_NODE_EDGES];
    uint16_t* incoming_edge_count = &incoming_edge_count_d[block_idx * CUDAPOA_MAX_NODES_PER_WINDOW];
    uint16_t* outoing_edges = &outgoing_edges_d[block_idx * CUDAPOA_MAX_NODES_PER_WINDOW * CUDAPOA_MAX_NODE_EDGES];
    uint16_t* outgoing_edge_count = &outgoing_edge_count_d[block_idx * CUDAPOA_MAX_NODES_PER_WINDOW];
    uint16_t* incoming_edge_weights = &incoming_edge_w_d[block_idx * CUDAPOA_MAX_NODES_PER_WINDOW * CUDAPOA_MAX_NODE_EDGES];
    uint16_t* outgoing_edge_weights = &outgoing_edge_w_d[block_idx * CUDAPOA_MAX_NODES_PER_WINDOW * CUDAPOA_MAX_NODE_EDGES];
    uint16_t* sorted_poa = &sorted_poa_d[block_idx * CUDAPOA_MAX_NODES_PER_WINDOW];
    uint16_t* node_id_to_pos = &node_id_to_pos_d[block_idx * CUDAPOA_MAX_NODES_PER_WINDOW];
    uint16_t* node_alignments = &node_alignments_d[block_idx * CUDAPOA_MAX_NODES_PER_WINDOW * CUDAPOA_MAX_NODE_ALIGNMENTS];
    uint16_t* node_alignment_count = &node_alignment_count_d[block_idx * CUDAPOA_MAX_NODES_PER_WINDOW];
    uint16_t* sorted_poa_local_edge_count = &sorted_poa_local_edge_count_d[block_idx * CUDAPOA_MAX_NODES_PER_WINDOW];

    int16_t* scores = &scores_d[CUDAPOA_MAX_MATRIX_GRAPH_DIMENSION * CUDAPOA_MAX_MATRIX_SEQUENCE_DIMENSION * block_idx];
    int16_t* alignment_graph = &alignment_graph_d[CUDAPOA_MAX_MATRIX_GRAPH_DIMENSION * block_idx];
    int16_t* alignment_read = &alignment_read_d[CUDAPOA_MAX_MATRIX_GRAPH_DIMENSION * block_idx];
    uint16_t* node_coverage_counts = &node_coverage_counts_d_[CUDAPOA_MAX_NODES_PER_WINDOW * block_idx];

    uint8_t* node_marks = &node_marks_d_[CUDAPOA_MAX_NODES_PER_WINDOW * block_idx];
    bool* check_aligned_nodes = &check_aligned_nodes_d_[CUDAPOA_MAX_NODES_PER_WINDOW * block_idx];
    uint16_t* nodes_to_visit = &nodes_to_visit_d_[CUDAPOA_MAX_NODES_PER_WINDOW * block_idx];

    //get Block-specific variables
    uint32_t window_idx = blockIdx.x;

    uint16_t * sequence_lengths = &sequence_lengths_d[window_details_d[window_idx].seq_len_buffer_offset];

    uint32_t num_sequences = window_details_d[window_idx].num_seqs;
    uint8_t * sequence = &sequences_d[window_details_d[window_idx].seq_starts];

    long long int t0 = clock64();

    if (thread_idx == 0)
    {

        // Create backbone for window based on first sequence in window.
        nodes[0] = sequence[0];
        sorted_poa[0] = 0;
        incoming_edge_count[0] = 0;
        node_alignment_count[0] = 0;
        node_id_to_pos[0] = 0;
        outgoing_edge_count[sequence_lengths[0] - 1] = 0;
        incoming_edge_weights[0] = 0;
        node_coverage_counts[0] = 1;

        //Build the rest of the graphs
        for (uint16_t nucleotide_idx=1; nucleotide_idx<sequence_lengths[0]; nucleotide_idx++){
            nodes[nucleotide_idx] = sequence[nucleotide_idx];
            sorted_poa[nucleotide_idx] = nucleotide_idx;
            outoing_edges[(nucleotide_idx-1) * CUDAPOA_MAX_NODE_EDGES] = nucleotide_idx;
            outgoing_edge_count[nucleotide_idx-1] = 1;
            incoming_edges[nucleotide_idx * CUDAPOA_MAX_NODE_EDGES] = nucleotide_idx - uint16_t(1);
            incoming_edge_weights[nucleotide_idx * CUDAPOA_MAX_NODE_EDGES] = 0;
            incoming_edge_count[nucleotide_idx] = 1;
            node_alignment_count[nucleotide_idx] = 0;
            node_id_to_pos[nucleotide_idx] = nucleotide_idx;
            node_coverage_counts[nucleotide_idx] = 1;
        }

    }

    __syncthreads();

    back_time += (clock64() - t0);

    //for(uint16_t i = 0; i < sequence_lengths[0]; i++)
    //{
    //    printf("%c ", nodes[i]);
    //}

    // Generate consensus only if sequences are aligned to graph.
    bool generate_consensus = false;

    //printf("window id %d, sequence %d\n", block_idx, num_sequences_in_window - 1);

    // Align each subsequent read, add alignment to graph, run topoligical sort.
    for(uint16_t s = 1; s < num_sequences; s++)
    {
        //printf("running window %d seq %d / %d\n", block_idx, s, num_sequences_in_window);
        uint16_t seq_len = sequence_lengths[s];
        sequence += sequence_lengths[s - 1]; // increment the pointer so it is pointing to correct sequence data
/*
        if (thread_idx == 0)
            printf("seq len is %i for sequence %i\n", seq_len, s);
*/

        //for(uint16_t i = 0; i < seq_len; i++)
        //{
        //    printf("%c ", seq[i]);
        //}

        //return;
        // Run DP step and fetch traceback.
        //bool found_node = false;
        //for(uint16_t i = 0; i < sequence_length_data[0]; i++)
        //{
        //    if (outgoing_edge_count[i] == 0)
        //    {
        //        printf("node %d has 0 oe\n", i);
        //        found_node = true;
        //    }
        //}
        //if (!found_node)
        //{
        //    printf("DID NOT FIND A NODE WITH NO OUTGOING EDGE before alignment!!!!\n");
        //    return;
        //}

        // print sorted graph
        //for(uint16_t i = 0; i < sequence_length_data[0]; i++)
        //{
        //    printf("%d ", sorted_poa[i]);
        //}
        //printf("\n");

        if (thread_idx == 0)
        {

            if (sequence_lengths[0] >= CUDAPOA_MAX_NODES_PER_WINDOW)
            {
                printf("Node count %d is greater than max matrix size %d\n", sequence_lengths[0], CUDAPOA_MAX_NODES_PER_WINDOW);
                return;
            }
            if (seq_len >= CUDAPOA_MAX_NODES_PER_WINDOW)
            {
                printf("Sequence len %d is greater than max matrix size %d\n", seq_len, CUDAPOA_MAX_NODES_PER_WINDOW);
                return;
            }


        }
        long long int start = clock64();

        // Run Needleman-Wunsch alignment between graph and new sequence.
/*
        if (thread_idx ==0)
            printf("running nw with sequence length of %i and sequence of %c %c %c %c %c\n", seq_len, sequence[0], sequence[1], sequence[2], sequence[3], sequence[4]);
*/

        uint16_t alignment_length = runNeedlemanWunsch(nodes,
                sorted_poa,
                node_id_to_pos,
                sequence_lengths[0],
                incoming_edge_count,
                incoming_edges,
                outgoing_edge_count,
                outoing_edges,
                sequence,
                seq_len,
                scores,
                alignment_graph,
                alignment_read);

        long long int nw_end = clock64();
        nw_time += (nw_end - start);

        __syncthreads();

        //found_node = false;
        //for(uint16_t i = 0; i < sequence_length_data[0]; i++)
        //{
        //    if (outgoing_edge_count[i] == 0)
        //    {
        //        printf("node %d has 0 oe\n", i);
        //        found_node = true;
        //    }
        //}
        //if (!found_node)
        //{
        //    printf("DID NOT FIND A NODE WITH NO OUTGOING EDGE before addition!!!!\n");
        //    return;
        //}

        start = clock64();

        if (thread_idx == 0)
        {

            // Add alignment to graph.
            //printf("running add\n");
            sequence_lengths[0] = addAlignmentToGraph(nodes, sequence_lengths[0],
                    node_alignments, node_alignment_count,
                    incoming_edges, incoming_edge_count,
                    outoing_edges, outgoing_edge_count,
                    incoming_edge_weights, outgoing_edge_weights,
                    alignment_length,
                    sorted_poa, alignment_graph, 
                    sequence, alignment_read,
                    node_coverage_counts);

            long long int add_end = clock64();
            add_time += (add_end - start);

            // Verify that each graph has at least one node with no outgoing edges.
            //bool found_node = false;
            //for(uint16_t i = 0; i < sequence_length_data[0]; i++)
            //{
            //    //printf("node id %d ie %d oe %d\n ", i, incoming_edge_count[i], outgoing_edge_count[i]);
            //    if (outgoing_edge_count[i] == 0)
            //        found_node = true;
            //}
            //if (!found_node)
            //{
            //    printf("DID NOT FIND A NODE WITH NO OUTGOING EDGE after addition!!!!\n");
            //    return;
            //}


            // Run a topsort on the graph. Not strictly necessary at this point
            //printf("running topsort\n");
#ifdef SPOA_ACCURATE
            // Exactly matches racon CPU results
            raconTopologicalSortDeviceUtil(sorted_poa,
                                      node_id_to_pos,
                                      sequence_lengths[0],
                                      incoming_edge_count,
                                      incoming_edges,
                                      node_alignment_count,
                                      node_alignments,
                                      node_marks,
                                      check_aligned_nodes,
                                      nodes_to_visit);
#else
            // Faster top sort
            topologicalSortDeviceUtil(sorted_poa,
                                      node_id_to_pos,
                                      sequence_lengths[0],
                                      incoming_edge_count,
                                      outoing_edges,
                                      outgoing_edge_count,
                                      sorted_poa_local_edge_count);
#endif

            long long int top_end = clock64();
            top_time += (top_end - add_end);
            //printf("done loop\n");
        }

        __syncthreads();

        generate_consensus = true;
    }

    // Dummy kernel code to copy first sequence as output.
    //uint8_t *input_row = &sequences_d[input_row_idx * sequences_pitch];
    //uint8_t *output_row = &consensus_d[block_idx * consensus_pitch];
    //for(uint32_t c = 0; c < sequence_lengths_d[block_idx * max_depth_per_window]; c++)
    //{
    //    output_row[c] = input_row[c];
    //}

    long long int consensus_time = 0;

    if (thread_idx == 0 && generate_consensus)
    {
        uint8_t* consensus = &consensus_d[block_idx * CUDAPOA_MAX_SEQUENCE_SIZE];
        uint16_t* coverage = &coverage_d[block_idx * CUDAPOA_MAX_SEQUENCE_SIZE];
        int32_t* consensus_scores = &consensus_scores_d[block_idx * CUDAPOA_MAX_NODES_PER_WINDOW];
        int16_t* consensus_predecessors = &consensus_predecessors_d[block_idx * CUDAPOA_MAX_NODES_PER_WINDOW];

        long long int start = clock64();
        generateConsensus(nodes,
                sequence_lengths[0],
                sorted_poa,
                node_id_to_pos,
                incoming_edges,
                incoming_edge_count,
                outoing_edges,
                outgoing_edge_count,
                incoming_edge_weights,
                consensus_predecessors,
                consensus_scores,
                consensus,
                coverage,
                node_coverage_counts,
                node_alignments, node_alignment_count);
        consensus_time = (clock64() - start);
    }
    //if (thread_idx == 0)
    //{
    //    long long int total = back_time + nw_time + add_time + top_time + consensus_time;
    //    printf("Total time of backbone generation is %lf %\n", ((double)back_time / total) * 100.f);
    //    printf("Total time of nw is %lf %\n", ((double)nw_time / total) * 100.f);
    //    printf("Total time of addition is %lf %\n", ((double)add_time / total) * 100.f);
    //    printf("Total time of topsort is %lf %\n", ((double)top_time / total) * 100.f);
    //    printf("Total time of consensus is %lf %\n", ((double)consensus_time / total) * 100.f);
    //}

}

// Host function call for POA kernel.
void generatePOA(uint8_t* consensus_d,
                 uint16_t* coverage_d,
                 uint8_t* sequences_d,
                 uint16_t * sequence_lengths_d,
                 nvidia::cudapoa::WindowDetails * window_details_d,
                 uint32_t total_windows,
                 uint32_t num_threads,
                 uint32_t num_blocks,
                 hipStream_t stream,
                 int16_t* scores,
                 int16_t* alignment_graph,
                 int16_t* alignment_read,
                 uint8_t* nodes,
                 uint16_t* incoming_edges,
                 uint16_t* incoming_edge_count,
                 uint16_t* outgoing_edges,
                 uint16_t* outgoing_edge_count,
                 uint16_t* incoming_edge_w,
                 uint16_t* outgoing_edge_w,
                 uint16_t* sorted_poa,
                 uint16_t* node_id_to_pos,
                 uint16_t* node_alignments,
                 uint16_t* node_alignment_count,
                 uint16_t* sorted_poa_local_edge_count,
                 int32_t* consensus_scores,
                 int16_t* consensus_predecessors,
                 uint8_t* node_marks,
                 bool* check_aligned_nodes,
                 uint16_t* nodes_to_visit,
                 uint16_t* node_coverage_counts)
{
    generatePOAKernel<<<num_blocks, num_threads, 0, stream>>>(consensus_d,
                                                              coverage_d,
                                                              sequences_d,
                                                              sequence_lengths_d,
                                                              window_details_d,
                                                              total_windows,
                                                              scores,
                                                              alignment_graph,
                                                              alignment_read,
                                                              nodes,
                                                              incoming_edges,
                                                              incoming_edge_count,
                                                              outgoing_edges,
                                                              outgoing_edge_count,
                                                              incoming_edge_w,
                                                              outgoing_edge_w,
                                                              sorted_poa,
                                                              node_id_to_pos,
                                                              node_alignments,
                                                              node_alignment_count,
                                                              sorted_poa_local_edge_count,
                                                              consensus_scores,
                                                              consensus_predecessors,
                                                              node_marks,
                                                              check_aligned_nodes,
                                                              nodes_to_visit,
                                                              node_coverage_counts);
}

} // namespace cudapoa

} // namespace nvidia
