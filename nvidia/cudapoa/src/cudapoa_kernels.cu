#include "hip/hip_runtime.h"
// Implementation file for CUDA POA kernels.

#include "cudapoa_kernels.cuh"
#include <stdio.h>

namespace nvidia {

namespace cudapoa {

const uint32_t MAX_DIMENSION = CUDAPOA_MAX_NODES_PER_WINDOW + 1;

// Device function for running topoligical sort on graph.
__device__
void topologicalSortDeviceUtil(uint16_t* sorted_poa,
                               uint16_t node_count,
                               uint16_t* incoming_edge_count,
                               uint16_t* outgoing_edges,
                               uint16_t* outgoing_edge_count)
{
    // Clear the incoming edge count for each node.
    uint16_t local_incoming_edge_count[CUDAPOA_MAX_NODES_PER_WINDOW];
    memset(local_incoming_edge_count, -1, 1000);
    uint16_t sorted_poa_position = 0;

    // Iterate through node IDs (since nodes are from 0
    // through node_count -1, a simple loop works) and fill 
    // out the incoming edge count.
    for(uint16_t n = 0; n < node_count; n++)
    {
        local_incoming_edge_count[n] = incoming_edge_count[n];
        // If we find a node ID has 0 incoming edges, add it to sorted nodes list.
        if (local_incoming_edge_count[n] == 0)
        {
            sorted_poa[sorted_poa_position++] = n;
        }
    }

    // Loop through set of node IDs with no incoming edges,
    // then iterate through their children. For each child decrement their 
    // incoming edge count. If incoming edge count of child == 0, 
    // add its node ID to the sorted order list.
    for(uint16_t n = 0; n < sorted_poa_position; n++)
    {
        uint16_t node = sorted_poa[n];
        for(uint16_t edge = 0; edge < outgoing_edge_count[node]; edge++)
        {
            uint16_t out_node = outgoing_edges[node * CUDAPOA_MAX_NODE_EDGES + edge];
            //printf("%d\n", out_node);
            local_incoming_edge_count[out_node]--;
            if (local_incoming_edge_count[out_node] == 0)
            {
                sorted_poa[sorted_poa_position++] = out_node;
            }
        }
    }

    // sorted_poa will have final ordering of node IDs.
}

// Device function for running Needleman-Wunsch dynamic programming loop.
__device__
void runNeedlemanWunsch(uint8_t* nodes,
                        uint16_t* graph,
                        uint16_t graph_count,
                        uint16_t* incoming_edge_count,
                        uint16_t* incoming_edges,
                        uint16_t* outgoing_edge_count,
                        uint16_t* outgoing_edges,
                        uint8_t* read,
                        uint16_t read_count,
                        int32_t* scores,
                        int16_t* traceback_i,
                        int16_t* traceback_j)
{
    // Assuming gap/mismatch penalty of 1, match rewards of -1.

    // Init boundary conditions.
    for(uint16_t i = 1; i < graph_count + 1; i++)
    {
        scores[i * MAX_DIMENSION + 0] = i;
    }
    for(uint16_t j = 1; j < read_count + 1; j++)
    {
        scores[0 * MAX_DIMENSION + j] = j;
    }

    // Run DP loop.
    for(uint16_t i = 1; i < graph_count + 1; i++)
    {
        for(uint16_t j = 1; j < read_count + 1; j++)
        {
            int32_t hor_val = scores[i * MAX_DIMENSION + (j-1)] + 1;
            int32_t ver_val = scores[(i-1) * MAX_DIMENSION + j] + 1;
            int32_t cell_val = (nodes[graph[i]] == read[j] ? -1 : 1);
            int32_t diag_val = scores[(i-1) * MAX_DIMENSION + (j-1)] + cell_val;

            int32_t final_val = min(hor_val, min(diag_val, ver_val));
            scores[i * MAX_DIMENSION + j] = final_val;
            if (hor_val <= final_val)
            {
                // Insert horizontal.
                traceback_i[i * MAX_DIMENSION + j] = i;
                traceback_j[i * MAX_DIMENSION + j] = j - 1;
            }
            else if (ver_val <= final_val)
            {
                // Insert vertical.
                traceback_i[i * MAX_DIMENSION + j] = i - 1;
                traceback_j[i * MAX_DIMENSION + j] = j;
            }
            else
            {
                // Insert diagonal.
                traceback_i[i * MAX_DIMENSION + j] = i - 1;
                traceback_j[i * MAX_DIMENSION + j] = j - 1;
            }
        }
    }
    
}

// Kernel for running POA.
__global__
void generatePOAKernel(uint8_t* consensus_d,
                       size_t consensus_pitch,
                       uint8_t* sequences_d,
                       size_t sequences_pitch,
                       uint32_t max_sequence_size,
                       uint16_t * num_sequences_per_window_d,
                       uint16_t * sequence_lengths_d,
                       uint32_t max_depth_per_window,
                       uint32_t total_windows,
                       int32_t* scores_d, int16_t* traceback_i_d, int16_t* traceback_j_d,
                       uint8_t* nodes_d,  uint16_t* incoming_edges_d, uint16_t* incoming_edge_count_d,
                       uint16_t* outgoing_edges_d, uint16_t* outgoing_edge_count_d,
                       uint16_t* incoming_edge_w_d, uint16_t* outgoing_edge_w_d,
                       uint16_t* sorted_poa_d)
{
    uint32_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_idx > total_windows)
        return;

    for(uint32_t window_idx = thread_idx; window_idx < total_windows; window_idx += blockDim.x)
    {
        // Memory layout for graph in adjacency list format.
        uint8_t* nodes = &nodes_d[CUDAPOA_MAX_NODES_PER_WINDOW * thread_idx];
        uint16_t* incoming_edges = &incoming_edges_d[thread_idx * CUDAPOA_MAX_NODES_PER_WINDOW * CUDAPOA_MAX_NODE_EDGES];
        uint16_t* incoming_edge_count = &incoming_edge_count_d[thread_idx * CUDAPOA_MAX_NODES_PER_WINDOW];
        uint16_t* outoing_edges = &outgoing_edges_d[thread_idx * CUDAPOA_MAX_NODES_PER_WINDOW * CUDAPOA_MAX_NODE_EDGES];
        uint16_t* outgoing_edge_count = &outgoing_edge_count_d[thread_idx * CUDAPOA_MAX_NODES_PER_WINDOW];
        uint16_t* incoming_edges_weights = &incoming_edge_w_d[thread_idx * CUDAPOA_MAX_NODES_PER_WINDOW * CUDAPOA_MAX_NODE_EDGES];
        uint16_t* outoing_edges_weights = &outgoing_edge_w_d[thread_idx * CUDAPOA_MAX_NODES_PER_WINDOW * CUDAPOA_MAX_NODE_EDGES];
        uint16_t* sorted_poa = &sorted_poa_d[thread_idx * CUDAPOA_MAX_NODES_PER_WINDOW];

        int32_t* scores = &scores_d[MAX_DIMENSION * MAX_DIMENSION * thread_idx];
        int16_t* traceback_i = &traceback_i_d[MAX_DIMENSION * MAX_DIMENSION * thread_idx];
        int16_t* traceback_j = &traceback_j_d[MAX_DIMENSION * MAX_DIMENSION * thread_idx];

        // Fetch the sequence data and sequence length sub-arrays for specific window ID.
        uint32_t input_row_idx = window_idx * max_depth_per_window;
        uint8_t* window_data = &sequences_d[input_row_idx * sequences_pitch];

        uint8_t num_sequences_in_window = num_sequences_per_window_d[window_idx];
        uint16_t* sequence_length_data = &sequence_lengths_d[window_idx * max_depth_per_window];

        // Create backbone for window based on first sequence in window.
        uint16_t node_count = 0;
        uint16_t sequence_0_length = sequence_length_data[0];
        nodes[0] = window_data[0];
        node_count++;
        sorted_poa[0] = 0;
        //Build the rest of the graphs
        for (int nucleotide_idx=1; nucleotide_idx<sequence_0_length; nucleotide_idx++){
            nodes[nucleotide_idx] = window_data[nucleotide_idx];
            node_count++;
            sorted_poa[nucleotide_idx] = nucleotide_idx;
            outoing_edges[(nucleotide_idx-1) * CUDAPOA_MAX_NODE_EDGES] = nucleotide_idx;
            outgoing_edge_count[nucleotide_idx-1] = 1;
            incoming_edges[nucleotide_idx * CUDAPOA_MAX_NODE_EDGES] = nucleotide_idx - 1;
            incoming_edge_count[nucleotide_idx] = 1;
        }

        //printf("node count %d\n", node_count);

        //// Align each subsequent read, add alignment to graph, run topoligical sort.
        for(uint16_t s = 1; s < num_sequences_in_window; s++)
        {
            uint8_t* seq = &window_data[s * max_sequence_size];
            uint16_t seq_len = sequence_length_data[s];

            // Run DP step and fetch traceback.
            runNeedlemanWunsch(nodes,
                               sorted_poa,
                               node_count,
                               incoming_edge_count,
                               incoming_edges,
                               outgoing_edge_count,
                               outoing_edges,
                               seq,
                               seq_len,
                               scores,
                               traceback_i,
                               traceback_j);

            // Fetch trackback alignment.

            // Add alignment to graph.

            // Run a topsort on the graph. Not strictly necessary at this point
            topologicalSortDeviceUtil(sorted_poa,
                                      node_count,
                                      incoming_edge_count,
                                      outoing_edges, outgoing_edge_count);
        }

        // Dummy kernel code to copy first sequence as output.
        uint8_t *input_row = &sequences_d[input_row_idx * sequences_pitch];
        uint8_t *output_row = &consensus_d[window_idx * consensus_pitch];
        for(uint32_t c = 0; c < sequence_lengths_d[window_idx * max_depth_per_window]; c++)
        {
            output_row[c] = input_row[c];
        }
    }

}

// Host function call for POA kernel.
void generatePOA(uint8_t* consensus_d,
                 size_t consensus_pitch,
                 uint8_t* sequences_d,
                 size_t sequences_pitch,
                 uint32_t max_sequence_size,
                 uint16_t* num_sequences_per_window_d,
                 uint16_t * sequence_lengths_d,
                 uint32_t max_depth_per_window,
                 uint32_t total_windows,
                 uint32_t num_threads, uint32_t num_blocks, hipStream_t stream,
                 int32_t* scores, int16_t* traceback_i, int16_t* traceback_j,
                 uint8_t* nodes,  uint16_t* incoming_edges, uint16_t* incoming_edge_count,
                 uint16_t* outgoing_edges, uint16_t* outgoing_edge_count,
                 uint16_t* incoming_edge_w, uint16_t* outgoing_edge_w,
                 uint16_t* sorted_poa)
{
    generatePOAKernel<<<num_blocks, num_threads, 0, stream>>>(consensus_d,
                                                              consensus_pitch,
                                                              sequences_d,
                                                              sequences_pitch,
                                                              max_sequence_size,
                                                              num_sequences_per_window_d,
                                                              sequence_lengths_d,
                                                              max_depth_per_window,
                                                              total_windows,
                                                              scores, traceback_i, traceback_j,
                                                              nodes, incoming_edges, incoming_edge_count,
                                                              outgoing_edges, outgoing_edge_count,
                                                              incoming_edge_w, outgoing_edge_w,
                                                              sorted_poa);
}


// Kernel for running topological independently.
__global__
void topologicalSortKernel(uint16_t* sorted_poa_d,
                           uint16_t node_count,
                           uint16_t* incoming_edge_count_d,
                           uint16_t* outgoing_edges_d,
                           uint16_t* outgoing_edge_count_d)
{
    if (blockIdx.x == 0 && threadIdx.x == 0)
    {
        topologicalSortDeviceUtil(sorted_poa_d,
                                  node_count,
                                  incoming_edge_count_d,
                                  outgoing_edges_d,
                                  outgoing_edge_count_d);
    }
}

// Host function for running topological sort kernel.
void topologicalSort(uint16_t* sorted_poa_d,
                     uint16_t node_count,
                     uint16_t* incoming_edge_count_d,
                     uint16_t* outgoing_edges_d,
                     uint16_t* outgoing_edge_count_d,
                     uint32_t num_threads, uint32_t num_blocks, hipStream_t stream)
{
    topologicalSortKernel<<<num_blocks, num_threads, 0, stream>>>(sorted_poa_d,
                                                                  node_count,
                                                                  incoming_edge_count_d,
                                                                  outgoing_edges_d,
                                                                  outgoing_edge_count_d);
}

} // namespace cudapoa

} // namespace nvidia
